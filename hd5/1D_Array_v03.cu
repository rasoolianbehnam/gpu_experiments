#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

void poisson_solve(int imax, int jmax, int kmax, int n1, int n2, int n3, int N, int iterations, float* V, float* g, float *R, float w, float h) {
      for (int kk=0; kk<iterations; kk++) {
        for (int I = 0; I < N; I ++) {
             int k = I % n3;
             int s1 = (I - k) / n3;
             int j = s1 % n2;
             int i = (s1 - j) / n2;
            if (i * j * k == 0 || i >= imax-1 || j >= jmax-1 || k >= kmax-1) continue;
            R[k + n3 * (j + n2 * (i))]=
                (V[k + n3 * (j + n2 * (i+1))]+
                     V[k + n3 * (j + n2 * (i-1))]+
                     V[k + n3 * (j+1 + n2 * (i))]+
                     V[k + n3 * (j-1 + n2 * (i))]+
                     V[k+1 + n3 * (j + n2 * (i))]+
                     V[k-1 + n3 * (j + n2 * (i))]
                 ) / 6.0 - V[k + n3 * (j + n2 * (i))]- (h*h)*g[k + n3 * (j + n2 * (i))]/6.0;
            V[k + n3 * (j + n2 * (i))] += w*R[k + n3 * (j + n2 * (i))];
        }
    }
}

__global__ void poisson_solve_1it_cu(int imax, int jmax, int kmax, int n1, int n2, int n3, int N, int iterations, float* V, float* g, float *R, float w, float h, int oddEven) {
    int index_x = threadIdx.x + blockDim.x * blockIdx.x;
    int stride_x = blockDim.x * gridDim.x;
        for (int I = index_x; I < N; I +=stride_x) {
             int k = I % n3;
             int s1 = (I - k) / n3;
             int j = s1 % n2;
             int i = (s1 - j) / n2;
            if (i * j * k == 0 || i >= imax-1 || j >= jmax-1 || k >= kmax-1) continue;
            if ((i+j+k)%2==oddEven) continue;
            R[k + n3 * (j + n2 * (i))]=
                (V[k + n3 * (j + n2 * (i+1))]+
                     V[k + n3 * (j + n2 * (i-1))]+
                     V[k + n3 * (j+1 + n2 * (i))]+
                     V[k + n3 * (j-1 + n2 * (i))]+
                     V[k+1 + n3 * (j + n2 * (i))]+
                     V[k-1 + n3 * (j + n2 * (i))]
                 ) / 6.0 - V[k + n3 * (j + n2 * (i))]- (h*h)*g[k + n3 * (j + n2 * (i))]/6.0;
            V[k + n3 * (j + n2 * (i))] += w*R[k + n3 * (j + n2 * (i))];
        }
}

void before_poisson(int imax, int jmax, int kmax, int tmax, float *ne, float *ni, float *difxne, float *difyne, float *difxni, float *difyni, float *difxyne, float *difxyni, float *Exy, float *fexy, float *fixy, float *g, float* g_temp, float *R, float *Ex, float *Ey, float *fex, float *fey, float *fix, float *fiy, float *V, float *L, float *difzne, float *difzni, float *Ez, float *fez, float *fiz, float qi, float qe, float kr, float ki, float si, float sf, float alpha, float q, float pie, float Ta , float w , float eps0 , float Te, float Ti, float B, float Kb, float me, float mi, float nue, float nui, float denominator_e, float denominator_i, float nn, float dt, float h, float wce, float wci, float mue, float mui, float dife, float difi) {
    int index_x = 0;
    int stride_x = 1;
    int  n1=imax+3, n2 = jmax+3, n3 = kmax+3,i,j,k,myTime,kk,I,N,s1; 
    N=n1*n2*n3;
        for (int I=0; I<N; I++) g_temp[I] = w*h*h*g[I]/6.;
        for (int I=0; I<N; I++) {
            int k = I % n3;
            int s1 = (I - k) / n3;
            int j = s1 % n2;
            int i = (s1 - j) / n2;
            if (i >= 1 && i < imax-1 
                    && j >= 1 && j < jmax-1 
                    && k >= 1 && k < kmax-1) {
                g_temp[I] += w/6.*(g_temp[I-1]+g_temp[I-n3]+g_temp[I-n3*n2]);
            }
            else {
                g_temp[I] = 0;
            }
        }

      for ( I = index_x; I < N; I += stride_x) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
        if (i * j * k == 0 || i >= imax-1 || j >= jmax-1 || k >= kmax-1) continue;
        g[k + n3 * (j + n2 * (i))]=-(ne[k + n3 * (j + n2 * (i))]*qe+ni[k + n3 * (j + n2 * (i))]*qi)/eps0;
    }
 
}
__global__ void before_poisson_cu(int imax, int jmax, int kmax, int tmax, float *ne, float *ni, float *difxne, float *difyne, float *difxni, float *difyni, float *difxyne, float *difxyni, float *Exy, float *fexy, float *fixy, float *g, float* g_temp, float *R, float *Ex, float *Ey, float *fex, float *fey, float *fix, float *fiy, float *V, float *L, float *difzne, float *difzni, float *Ez, float *fez, float *fiz, float qi, float qe, float kr, float ki, float si, float sf, float alpha, float q, float pie, float Ta , float w , float eps0 , float Te, float Ti, float B, float Kb, float me, float mi, float nue, float nui, float denominator_e, float denominator_i, float nn, float dt, float h, float wce, float wci, float mue, float mui, float dife, float difi) {
    int index_x = threadIdx.x + blockDim.x * blockIdx.x;
    int stride_x = blockDim.x * gridDim.x;
    int  n1=imax+3, n2 = jmax+3, n3 = kmax+3,i,j,k,myTime,kk,I,N,s1; 
    N=n1*n2*n3;
    for (int I=index_x; I<N; I+=stride_x) g_temp[I] = w*h*h*g[I]/6.;
    for (int I=index_x; I<N; I+=stride_x) {
        int k = I % n3;
        int s1 = (I - k) / n3;
        int j = s1 % n2;
        int i = (s1 - j) / n2;
        if (i >= 1 && i < imax-1 
                && j >= 1 && j < jmax-1 
                && k >= 1 && k < kmax-1) {
            g_temp[I] += w/6.*(g_temp[I-1]+g_temp[I-n3]+g_temp[I-n3*n2]);
        }
        else {
            g_temp[I] = 0;
        }
    }

      for ( I = index_x; I < N; I += stride_x) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
        if (i * j * k == 0 || i >= imax-1 || j >= jmax-1 || k >= kmax-1) continue;
        g[k + n3 * (j + n2 * (i))]=-(ne[k + n3 * (j + n2 * (i))]*qe+ni[k + n3 * (j + n2 * (i))]*qi)/eps0;
    }
 
}

void after_poisson(int imax, int jmax, int kmax, int tmax, float *ne, float *ni, float* ne_temp, float* ni_temp, float *difxne, float *difyne, float *difxni, float *difyni, float *difxyne, float *difxyni, float *Exy, float *fexy, float *fixy, float *g, float* g_temp, float *R, float *Ex, float *Ey, float *fex, float *fey, float *fix, float *fiy, float *V, float *L, float *difzne, float *difzni, float *Ez, float *fez, float *fiz, float qi, float qe, float kr, float ki, float si, float sf, float alpha, float q, float pie, float Ta , float w , float eps0 , float Te, float Ti, float B, float Kb, float me, float mi, float nue, float nui, float denominator_e, float denominator_i, float nn, float dt, float h, float wce, float wci, float mue, float mui, float dife, float difi) {
    int index_x = 0;
    int stride_x = 1;
    int  n1=imax+3, n2 = jmax+3, n3 = kmax+3,i,j,k,myTime,kk,I,N,s1; 
    N=n1*n2*n3;
  for ( I = index_x; I < N; I += stride_x) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
        if (i >= imax-1 || j >= jmax || k >= kmax) continue;
        Ex[k + n3 * (j + n2 * (i))]= (V[k + n3 * (j + n2 * (i))]-V[k + n3 * (j + n2 * (i+1))])/h;
        difxne[k + n3 * (j + n2 * (i))]=(ne[k + n3 * (j + n2 * (i+1))]-ne[k + n3 * (j + n2 * (i))])/h;
        difxni[k + n3 * (j + n2 * (i))]=(ni[k + n3 * (j + n2 * (i+1))]-ni[k + n3 * (j + n2 * (i))])/h;
        }


    for ( I = index_x; I < N; I += stride_x) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
        if (i >= imax || j >= jmax-1 || k >= kmax) continue;
        Ey[k + n3 * (j + n2 * (i))]= (V[k + n3 * (j + n2 * (i))]-V[k + n3 * (j+1 + n2 * (i))])/h;
        difyne[k + n3 * (j + n2 * (i))]=(ne[k + n3 * (j+1 + n2 * (i))]-ne[k + n3 * (j + n2 * (i))])/h;
        difyni[k + n3 * (j + n2 * (i))]=(ni[k + n3 * (j+1 + n2 * (i))]-ni[k + n3 * (j + n2 * (i))])/h;
        }


    for ( I = index_x; I < N; I += stride_x) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
        if (i >= imax || j >= jmax || k >= kmax-1) continue;
       Ez[k + n3 * (j + n2 * (i))]= (V[k + n3 * (j + n2 * (i))]-V[k+1 + n3 * (j + n2 * (i))])/h;
       difzne[k + n3 * (j + n2 * (i))]=(ne[k+1 + n3 * (j + n2 * (i))]-ne[k + n3 * (j + n2 * (i))])/h;
       difzni[k + n3 * (j + n2 * (i))]=(ni[k+1 + n3 * (j + n2 * (i))]-ni[k + n3 * (j + n2 * (i))])/h;
     }

// -----------------------------------------------------------------------------------------------
       /* Since I am using mid points for Calculating electric field and density gradient,
        to calculate Ex at any point that I don't have it directly, the average over
        the neighboring points is used instead. these average variables are, exy, fexy, fixy, ...*/
        // Calculating the average values of Ex and gradiant_x
   for ( I = index_x; I < N; I += stride_x) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;

        Exy[k + n3 * (j + n2 * (i))]= 0.0 ;
        difxyne[k + n3 * (j + n2 * (i))]=0.0;
        difxyni[k + n3 * (j + n2 * (i))]=0.0;
    }

    for ( I = index_x; I < N; I += stride_x) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
        if (i * k == 0 ||  i >= imax-1 || j >= jmax-1 || k >= kmax-1) continue;
        Exy[k + n3 * (j + n2 * (i))]= 0.25*(Ex[k + n3 * (j + n2 * (i))]+Ex[k + n3 * (j+1 + n2 * (i))]+Ex[k + n3 * (j + n2 * (i-1))]+Ex[k + n3 * (j+1 + n2 * (i-1))]) ;
        difxyne[k + n3 * (j + n2 * (i))]=0.25*(difxne[k + n3 * (j + n2 * (i))]+difxne[k + n3 * (j+1 + n2 * (i))]+difxne[k + n3 * (j + n2 * (i-1))]+difxne[k + n3 * (j+1 + n2 * (i-1))]);
        difxyni[k + n3 * (j + n2 * (i))]=0.25*(difxni[k + n3 * (j + n2 * (i))]+difxni[k + n3 * (j+1 + n2 * (i))]+difxni[k + n3 * (j + n2 * (i-1))]+difxni[k + n3 * (j+1 + n2 * (i-1))]);
    }


// -----------------------------------------------------------------------------------------------
        // Here we calculate the fluxes in y direction

       for ( I = index_x; I < N; I += stride_x) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
        if (i * k == 0 ||  i >= imax-1 || j >= jmax-1 || k >= kmax-1) continue;
        fey[k + n3 * (j + n2 * (i))]= (-0.5*(ne[k + n3 * (j+1 + n2 * (i))]+ne[k + n3 * (j + n2 * (i))])*mue*Ey[k + n3 * (j + n2 * (i))]-dife*difyne[k + n3 * (j + n2 * (i))]
        -wce*q*0.5*(ne[k + n3 * (j+1 + n2 * (i))]+ne[k + n3 * (j + n2 * (i))])*Exy[k + n3 * (j + n2 * (i))]/(me*nue*nue)-wce*dife*difxyne[k + n3 * (j + n2 * (i))]/nue)/denominator_e;
        fiy[k + n3 * (j + n2 * (i))]= (0.5*(ni[k + n3 * (j+1 + n2 * (i))]+ni[k + n3 * (j + n2 * (i))])*mui*Ey[k + n3 * (j + n2 * (i))]-difi*difyni[k + n3 * (j + n2 * (i))]
        -wci*q*0.5*(ni[k + n3 * (j+1 + n2 * (i))]+ni[k + n3 * (j + n2 * (i))])*Exy[k + n3 * (j + n2 * (i))]/(mi*nui*nui)+wci*difi*difxyni[k + n3 * (j + n2 * (i))]/nui)/denominator_i;
    }


    for ( I =index_x; I < n1 * n3; I += stride_x) {
         k = I % n3;
         i = (I - k) / n3;

        if (i * k == 0 ||  i >= imax-1 || k >= kmax-1) continue;

        if (fey[k + n3 * (0 + n2 * (i))] > 0.0){
                fey[k + n3 * (0 + n2 * (i))] = 0.0;
                }

        if (fiy[k + n3 * (0 + n2 * (i))] > 0.0){
                fiy[k + n3 * (0 + n2 * (i))] = 0.0;
                }

        if (fey[k + n3 * (jmax-2 + n2 * (i))] < 0.0){
                fey[k + n3 * (jmax-2 + n2 * (i))] = 0.0;
                }

        if (fiy[k + n3 * (jmax-2 + n2 * (i))] < 0.0){
                fiy[k + n3 * (jmax-2 + n2 * (i))] = 0.0;
                }

    }

// -----------------------------------------------------------------------------------------------
        // Calculating the average Exy and difxy to be used in x direction fluxes
       // Calculating the average values of Ey and gradiant_y


      for ( I = index_x; I < N; I += stride_x) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;

        Exy[k + n3 * (j + n2 * (i))]= 0.0 ;
        difxyne[k + n3 * (j + n2 * (i))]=0.0;
        difxyni[k + n3 * (j + n2 * (i))]=0.0;
    }


      for ( I = index_x; I < N; I += stride_x) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
        if (j * k == 0 || i >= imax-1 || j >= jmax-1 || k >= kmax-1) continue;
        Exy[k + n3 * (j + n2 * (i))]= 0.25*(Ey[k + n3 * (j + n2 * (i))]+Ey[k + n3 * (j-1 + n2 * (i))]+Ey[k + n3 * (j + n2 * (i+1))]+Ey[k + n3 * (j-1 + n2 * (i+1))]);
        difxyne[k + n3 * (j + n2 * (i))]= 0.25*(difyne[k + n3 * (j + n2 * (i))]+difyne[k + n3 * (j-1 + n2 * (i))]+difyne[k + n3 * (j + n2 * (i+1))]+difyne[k + n3 * (j-1 + n2 * (i+1))]);
        difxyni[k + n3 * (j + n2 * (i))]= 0.25*(difyni[k + n3 * (j + n2 * (i))]+difyni[k + n3 * (j-1 + n2 * (i))]+difyni[k + n3 * (j + n2 * (i+1))]+difyni[k + n3 * (j-1 + n2 * (i+1))]);

    }

// -----------------------------------------------------------------------------------------------
        // Now ready to calculate the fluxes in x direction

    for ( I = index_x; I < N; I += stride_x) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
        if (j * k == 0 || i >= imax-1 || j >= jmax-1 || k >= kmax-1) continue;
        fex[k + n3 * (j + n2 * (i))]=(-0.5*(ne[k + n3 * (j + n2 * (i))]+ne[k + n3 * (j + n2 * (i+1))])*mue*Ex[k + n3 * (j + n2 * (i))]-dife*difxne[k + n3 * (j + n2 * (i))]
        +wce*dife*difxyne[k + n3 * (j + n2 * (i))]/nue+wce*q*0.5*(ne[k + n3 * (j + n2 * (i))]+ne[k + n3 * (j + n2 * (i+1))])/(me*nue*nue)*Exy[k + n3 * (j + n2 * (i))])/denominator_e;
        fix[k + n3 * (j + n2 * (i))]=(0.5*(ni[k + n3 * (j + n2 * (i))]+ni[k + n3 * (j + n2 * (i+1))])*mui*Ex[k + n3 * (j + n2 * (i))]-difi*difxni[k + n3 * (j + n2 * (i))]
        -wci*difi*difxyni[k + n3 * (j + n2 * (i))]/nui+wci*q*0.5*(ni[k + n3 * (j + n2 * (i))]+ni[k + n3 * (j + n2 * (i+1))])*Exy[k + n3 * (j + n2 * (i))]/(mi*nui*nui))/denominator_i;
    }


    for ( I = index_x; I < n2 * n3; I += stride_x) {
         k = I % n3;
         j = (I - k) / n3;

        if (j * k == 0 ||  j >= jmax-1 || k >= kmax-1) continue;

        if (fex[k + n3 * (j + n2 * (0))] > 0.0){
                fex[k + n3 * (j + n2 * (0))] = 0.0;
                }

        if (fix[k + n3 * (j + n2 * (0))] > 0.0){
                fix[k + n3 * (j + n2 * (0))] = 0.0;
                }

        if (fex[k + n3 * (j + n2 * (imax-2))] < 0.0){
                fex[k + n3 * (j + n2 * (imax-2))] = 0.0;
                }

        if (fix[k + n3 * (j + n2 * (imax-2))] < 0.0){
                fix[k + n3 * (j + n2 * (imax-2))] = 0.0;
                }

    }

// -----------------------------------------------------------------------------------------------

        // Now we calculate the fluxes in z direction
      for ( I = index_x; I < N; I += stride_x) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
        if (i * j == 0 || i >= imax-1 || j >= jmax-1 || k >= kmax-1) continue;
        fez[k + n3 * (j + n2 * (i))]=-0.5*(ne[k + n3 * (j + n2 * (i))]+ne[k+1 + n3 * (j + n2 * (i))])*mue*Ez[k + n3 * (j + n2 * (i))]-dife*difzne[k + n3 * (j + n2 * (i))];
        fiz[k + n3 * (j + n2 * (i))]=0.5*(ni[k + n3 * (j + n2 * (i))]+ni[k+1 + n3 * (j + n2 * (i))])*mui*Ez[k + n3 * (j + n2 * (i))]-difi*difzni[k + n3 * (j + n2 * (i))];

    }
       // BC on fluxes

    for ( I = index_x; I < n1 * n2; I += stride_x) {
         j = I % n2;
         i = (I - j) / n2;
        if (i * j == 0 || i >= imax-1 || j >= jmax-1) continue;
        if (fez[0 + n3 * (j + n2 * (i))]>0.0){
            fez[0 + n3 * (j + n2 * (i))]=0.0;
        }
        if (fiz[0 + n3 * (j + n2 * (i))]>0.0){
            fiz[0 + n3 * (j + n2 * (i))]=0.0;
        }
        if (fez[kmax-2 + n3 * (j + n2 * (i))]<0.0){
            fez[kmax-2 + n3 * (j + n2 * (i))]=0.0;
        }
        if (fiz[kmax-2 + n3 * (j + n2 * (i))]<0.0){
            fiz[kmax-2 + n3 * (j + n2 * (i))]=0.0;
        }
    }
// -----------------------------------------------------------------------------------------------


       for ( I = index_x; I < N; I += stride_x) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
        if (i * j * k == 0 || i >= imax || j >= jmax || k >= kmax) continue;
        ne[k + n3 * (j + n2 * (i))]=ne[k + n3 * (j + n2 * (i))]-dt*(fex[k + n3 * (j + n2 * (i))]-fex[k + n3 * (j + n2 * (i-1))]+fey[k + n3 * (j + n2 * (i))]-fey[k + n3 * (j-1 + n2 * (i))]+fez[k + n3 * (j + n2 * (i))]-fez[k-1 + n3 * (j + n2 * (i))])/h ;
        ni[k + n3 * (j + n2 * (i))]=ni[k + n3 * (j + n2 * (i))]-dt*(fix[k + n3 * (j + n2 * (i))]-fix[k + n3 * (j + n2 * (i-1))]+fiy[k + n3 * (j + n2 * (i))]-fiy[k + n3 * (j-1 + n2 * (i))]+fiz[k + n3 * (j + n2 * (i))]-fiz[k-1 + n3 * (j + n2 * (i))])/h ;
    }





        for ( I = index_x; I < n1 * n2; I += stride_x) {
         j = I % n2;
         i = (I - j) / n2;
        if (i * j == 0 || i >= imax || j >= jmax ) continue;

        ne[0 + n3 * (j + n2 * (i))] = -dt*fez[0 + n3 * (j + n2 * (i))]/h ;
        ni[0 + n3 * (j + n2 * (i))] = -dt*fiz[0 + n3 * (j + n2 * (i))]/h ;

    }

     for ( I = index_x; I < n1 * n3; I += stride_x) {
         k = I % n3;
         i = (I - k) / n3;

        if (i * k == 0 ||  i >= imax || k >= kmax) continue;

        ne[k + n3 * (0 + n2 * (i))] = -dt*fey[k + n3 * (0 + n2 * (i))]/h ;
        ni[k + n3 * (0 + n2 * (i))] = -dt*fiy[k + n3 * (0 + n2 * (i))]/h ;

    }




      for ( I = index_x; I < n2 * n3; I += stride_x) {
         k = I % n3;
         j = (I - k) / n3;
        if (j * k == 0 ||  j >= jmax || k >= kmax) continue;

       ne[k + n3 * (j + n2 * (0))]= -dt*fex[k + n3 * (j + n2 * (0))]/h ;
       ni[k + n3 * (j + n2 * (0))]= -dt*fix[k + n3 * (j + n2 * (0))]/h ;


    }




        // BC on densities

     for ( I = index_x; I < n1 * n3; I += stride_x) {
         k = I % n3;
         i = (I - k) / n3;

        if (i * k == 0 ||  i >= imax || k >= kmax) continue;

        ne[k + n3 * (0 + n2 * (i))] = 0.0 ;
        ni[k + n3 * (0 + n2 * (i))] = 0.0 ;

        ne[k + n3 * (jmax-1 + n2 * (i))] = 0.0 ;
        ni[k + n3 * (jmax-1 + n2 * (i))] = 0.0 ;

    }




      for ( I = index_x; I < n2 * n3; I += stride_x) {
         k = I % n3;
         j = (I - k) / n3;
        if (j * k == 0 ||  j >= jmax || k >= kmax) continue;

       ne[k + n3 * (j + n2 * (0))]= 0.0 ;
       ni[k + n3 * (j + n2 * (0))]= 0.0 ;

       ne[k + n3 * (j + n2 * (imax-1))]= 0.0 ;
       ni[k + n3 * (j + n2 * (imax-1))]= 0.0 ;


    }



    for ( I = index_x; I < n1*n2; I += stride_x) {
         j = I % n2;
         i = (I - j) / n2;
        if (i * j == 0 || i >= imax+1 || j >= jmax+1) continue;
        ne[kmax-1 + n3 * (j + n2 * (i))]=0.0;
        ne[0 + n3 * (j + n2 * (i))]=0.0;
        ni[kmax-1 + n3 * (j + n2 * (i))]=0.0;
        ni[0 + n3 * (j + n2 * (i))]=0.0;

    }



        // calculating the loss
//         sf=0.0;
//       for ( I = index_x; I < N; I += stride_x) {
//         k = I % n3;
//         s1 = (I - k) / n3;
//         j = s1 % n2;
//         i = (s1 - j) / n2;
//         if (i * j * k == 0 || i >= imax || j >= jmax || k >= kmax) continue;
//        sf=sf+ne[k + n3 * (j + n2 * (i))] ;
//    }
//       printf("sf = %f\n", sf);
//
//        alpha=(si-sf)/sf;
//
//    for ( I = index_x; I < N; I += stride_x) {
//         k = I % n3;
//         s1 = (I - k) / n3;
//         j = s1 % n2;
//         i = (s1 - j) / n2;
//        if (i * j * k == 0 || i >= imax-1 || j >= jmax-1 || k >= kmax-1) continue;
//        ne_temp[k + n3 * (j + n2 * (i))]=ne[k + n3 * (j + n2 * (i))]+alpha*ne[k + n3 * (j + n2 * (i))] ;
//        ni_temp[k + n3 * (j + n2 * (i))]=ni[k + n3 * (j + n2 * (i))]+alpha*ne[k + n3 * (j + n2 * (i))] ;
//    }
//    for ( I = index_x; I < N; I += stride_x) {
//        ne[I] = ne_temp[I];
//        ni[I] = ni_temp[I];
//    }
      // if (myTime%100==0.0){
     //  }

}

__global__ void after_poisson_cu(int imax, int jmax, int kmax, int tmax, float *ne, float *ni, float *ne_temp, float* ni_temp, float *difxne, float *difyne, float *difxni, float *difyni, float *difxyne, float *difxyni, float *Exy, float *fexy, float *fixy, float *g, float* g_temp, float *R, float *Ex, float *Ey, float *fex, float *fey, float *fix, float *fiy, float *V, float *L, float *difzne, float *difzni, float *Ez, float *fez, float *fiz, float qi, float qe, float kr, float ki, float si, float sf, float alpha, float q, float pie, float Ta , float w , float eps0 , float Te, float Ti, float B, float Kb, float me, float mi, float nue, float nui, float denominator_e, float denominator_i, float nn, float dt, float h, float wce, float wci, float mue, float mui, float dife, float difi, float *sf_temp) {
    int index_x = threadIdx.x + blockDim.x * blockIdx.x;
    int stride_x = blockDim.x * gridDim.x;
    int  n1=imax+3, n2 = jmax+3, n3 = kmax+3,i,j,k,myTime,kk,I,N,s1; 
    N=n1*n2*n3;
  for ( I = index_x; I < N; I += stride_x) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
        if (i >= imax-1 || j >= jmax || k >= kmax) continue;
        Ex[k + n3 * (j + n2 * (i))]= (V[k + n3 * (j + n2 * (i))]-V[k + n3 * (j + n2 * (i+1))])/h;
        difxne[k + n3 * (j + n2 * (i))]=(ne[k + n3 * (j + n2 * (i+1))]-ne[k + n3 * (j + n2 * (i))])/h;
        difxni[k + n3 * (j + n2 * (i))]=(ni[k + n3 * (j + n2 * (i+1))]-ni[k + n3 * (j + n2 * (i))])/h;
        }


    for ( I = index_x; I < N; I += stride_x) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
        if (i >= imax || j >= jmax-1 || k >= kmax) continue;
        Ey[k + n3 * (j + n2 * (i))]= (V[k + n3 * (j + n2 * (i))]-V[k + n3 * (j+1 + n2 * (i))])/h;
        difyne[k + n3 * (j + n2 * (i))]=(ne[k + n3 * (j+1 + n2 * (i))]-ne[k + n3 * (j + n2 * (i))])/h;
        difyni[k + n3 * (j + n2 * (i))]=(ni[k + n3 * (j+1 + n2 * (i))]-ni[k + n3 * (j + n2 * (i))])/h;
        }


    for ( I = index_x; I < N; I += stride_x) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
        if (i >= imax || j >= jmax || k >= kmax-1) continue;
       Ez[k + n3 * (j + n2 * (i))]= (V[k + n3 * (j + n2 * (i))]-V[k+1 + n3 * (j + n2 * (i))])/h;
       difzne[k + n3 * (j + n2 * (i))]=(ne[k+1 + n3 * (j + n2 * (i))]-ne[k + n3 * (j + n2 * (i))])/h;
       difzni[k + n3 * (j + n2 * (i))]=(ni[k+1 + n3 * (j + n2 * (i))]-ni[k + n3 * (j + n2 * (i))])/h;
     }

// -----------------------------------------------------------------------------------------------
       /* Since I am using mid points for Calculating electric field and density gradient,
        to calculate Ex at any point that I don't have it directly, the average over
        the neighboring points is used instead. these average variables are, exy, fexy, fixy, ...*/
        // Calculating the average values of Ex and gradiant_x
   for ( I = index_x; I < N; I += stride_x) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;

        Exy[k + n3 * (j + n2 * (i))]= 0.0 ;
        difxyne[k + n3 * (j + n2 * (i))]=0.0;
        difxyni[k + n3 * (j + n2 * (i))]=0.0;
    }

    for ( I = index_x; I < N; I += stride_x) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
        if (i * k == 0 ||  i >= imax-1 || j >= jmax-1 || k >= kmax-1) continue;
        Exy[k + n3 * (j + n2 * (i))]= 0.25*(Ex[k + n3 * (j + n2 * (i))]+Ex[k + n3 * (j+1 + n2 * (i))]+Ex[k + n3 * (j + n2 * (i-1))]+Ex[k + n3 * (j+1 + n2 * (i-1))]) ;
        difxyne[k + n3 * (j + n2 * (i))]=0.25*(difxne[k + n3 * (j + n2 * (i))]+difxne[k + n3 * (j+1 + n2 * (i))]+difxne[k + n3 * (j + n2 * (i-1))]+difxne[k + n3 * (j+1 + n2 * (i-1))]);
        difxyni[k + n3 * (j + n2 * (i))]=0.25*(difxni[k + n3 * (j + n2 * (i))]+difxni[k + n3 * (j+1 + n2 * (i))]+difxni[k + n3 * (j + n2 * (i-1))]+difxni[k + n3 * (j+1 + n2 * (i-1))]);
    }


// -----------------------------------------------------------------------------------------------
        // Here we calculate the fluxes in y direction

       for ( I = index_x; I < N; I += stride_x) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
        if (i * k == 0 ||  i >= imax-1 || j >= jmax-1 || k >= kmax-1) continue;
        fey[k + n3 * (j + n2 * (i))]= (-0.5*(ne[k + n3 * (j+1 + n2 * (i))]+ne[k + n3 * (j + n2 * (i))])*mue*Ey[k + n3 * (j + n2 * (i))]-dife*difyne[k + n3 * (j + n2 * (i))]
        -wce*q*0.5*(ne[k + n3 * (j+1 + n2 * (i))]+ne[k + n3 * (j + n2 * (i))])*Exy[k + n3 * (j + n2 * (i))]/(me*nue*nue)-wce*dife*difxyne[k + n3 * (j + n2 * (i))]/nue)/denominator_e;
        fiy[k + n3 * (j + n2 * (i))]= (0.5*(ni[k + n3 * (j+1 + n2 * (i))]+ni[k + n3 * (j + n2 * (i))])*mui*Ey[k + n3 * (j + n2 * (i))]-difi*difyni[k + n3 * (j + n2 * (i))]
        -wci*q*0.5*(ni[k + n3 * (j+1 + n2 * (i))]+ni[k + n3 * (j + n2 * (i))])*Exy[k + n3 * (j + n2 * (i))]/(mi*nui*nui)+wci*difi*difxyni[k + n3 * (j + n2 * (i))]/nui)/denominator_i;
    }


    for ( I =index_x; I < n1 * n3; I += stride_x) {
         k = I % n3;
         i = (I - k) / n3;

        if (i * k == 0 ||  i >= imax-1 || k >= kmax-1) continue;

        if (fey[k + n3 * (0 + n2 * (i))] > 0.0){
                fey[k + n3 * (0 + n2 * (i))] = 0.0;
                }

        if (fiy[k + n3 * (0 + n2 * (i))] > 0.0){
                fiy[k + n3 * (0 + n2 * (i))] = 0.0;
                }

        if (fey[k + n3 * (jmax-2 + n2 * (i))] < 0.0){
                fey[k + n3 * (jmax-2 + n2 * (i))] = 0.0;
                }

        if (fiy[k + n3 * (jmax-2 + n2 * (i))] < 0.0){
                fiy[k + n3 * (jmax-2 + n2 * (i))] = 0.0;
                }

    }

// -----------------------------------------------------------------------------------------------
        // Calculating the average Exy and difxy to be used in x direction fluxes
       // Calculating the average values of Ey and gradiant_y


      for ( I = index_x; I < N; I += stride_x) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;

        Exy[k + n3 * (j + n2 * (i))]= 0.0 ;
        difxyne[k + n3 * (j + n2 * (i))]=0.0;
        difxyni[k + n3 * (j + n2 * (i))]=0.0;
    }


      for ( I = index_x; I < N; I += stride_x) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
        if (j * k == 0 || i >= imax-1 || j >= jmax-1 || k >= kmax-1) continue;
        Exy[k + n3 * (j + n2 * (i))]= 0.25*(Ey[k + n3 * (j + n2 * (i))]+Ey[k + n3 * (j-1 + n2 * (i))]+Ey[k + n3 * (j + n2 * (i+1))]+Ey[k + n3 * (j-1 + n2 * (i+1))]);
        difxyne[k + n3 * (j + n2 * (i))]= 0.25*(difyne[k + n3 * (j + n2 * (i))]+difyne[k + n3 * (j-1 + n2 * (i))]+difyne[k + n3 * (j + n2 * (i+1))]+difyne[k + n3 * (j-1 + n2 * (i+1))]);
        difxyni[k + n3 * (j + n2 * (i))]= 0.25*(difyni[k + n3 * (j + n2 * (i))]+difyni[k + n3 * (j-1 + n2 * (i))]+difyni[k + n3 * (j + n2 * (i+1))]+difyni[k + n3 * (j-1 + n2 * (i+1))]);

    }

// -----------------------------------------------------------------------------------------------
        // Now ready to calculate the fluxes in x direction

    for ( I = index_x; I < N; I += stride_x) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
        if (j * k == 0 || i >= imax-1 || j >= jmax-1 || k >= kmax-1) continue;
        fex[k + n3 * (j + n2 * (i))]=(-0.5*(ne[k + n3 * (j + n2 * (i))]+ne[k + n3 * (j + n2 * (i+1))])*mue*Ex[k + n3 * (j + n2 * (i))]-dife*difxne[k + n3 * (j + n2 * (i))]
        +wce*dife*difxyne[k + n3 * (j + n2 * (i))]/nue+wce*q*0.5*(ne[k + n3 * (j + n2 * (i))]+ne[k + n3 * (j + n2 * (i+1))])/(me*nue*nue)*Exy[k + n3 * (j + n2 * (i))])/denominator_e;
        fix[k + n3 * (j + n2 * (i))]=(0.5*(ni[k + n3 * (j + n2 * (i))]+ni[k + n3 * (j + n2 * (i+1))])*mui*Ex[k + n3 * (j + n2 * (i))]-difi*difxni[k + n3 * (j + n2 * (i))]
        -wci*difi*difxyni[k + n3 * (j + n2 * (i))]/nui+wci*q*0.5*(ni[k + n3 * (j + n2 * (i))]+ni[k + n3 * (j + n2 * (i+1))])*Exy[k + n3 * (j + n2 * (i))]/(mi*nui*nui))/denominator_i;
    }


    for ( I = index_x; I < n2 * n3; I += stride_x) {
         k = I % n3;
         j = (I - k) / n3;

        if (j * k == 0 ||  j >= jmax-1 || k >= kmax-1) continue;

        if (fex[k + n3 * (j + n2 * (0))] > 0.0){
                fex[k + n3 * (j + n2 * (0))] = 0.0;
                }

        if (fix[k + n3 * (j + n2 * (0))] > 0.0){
                fix[k + n3 * (j + n2 * (0))] = 0.0;
                }

        if (fex[k + n3 * (j + n2 * (imax-2))] < 0.0){
                fex[k + n3 * (j + n2 * (imax-2))] = 0.0;
                }

        if (fix[k + n3 * (j + n2 * (imax-2))] < 0.0){
                fix[k + n3 * (j + n2 * (imax-2))] = 0.0;
                }

    }

// -----------------------------------------------------------------------------------------------

        // Now we calculate the fluxes in z direction
      for ( I = index_x; I < N; I += stride_x) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
        if (i * j == 0 || i >= imax-1 || j >= jmax-1 || k >= kmax-1) continue;
        fez[k + n3 * (j + n2 * (i))]=-0.5*(ne[k + n3 * (j + n2 * (i))]+ne[k+1 + n3 * (j + n2 * (i))])*mue*Ez[k + n3 * (j + n2 * (i))]-dife*difzne[k + n3 * (j + n2 * (i))];
        fiz[k + n3 * (j + n2 * (i))]=0.5*(ni[k + n3 * (j + n2 * (i))]+ni[k+1 + n3 * (j + n2 * (i))])*mui*Ez[k + n3 * (j + n2 * (i))]-difi*difzni[k + n3 * (j + n2 * (i))];

    }
       // BC on fluxes

    for ( I = index_x; I < n1 * n2; I += stride_x) {
         j = I % n2;
         i = (I - j) / n2;
        if (i * j == 0 || i >= imax-1 || j >= jmax-1) continue;
        if (fez[0 + n3 * (j + n2 * (i))]>0.0){
            fez[0 + n3 * (j + n2 * (i))]=0.0;
        }
        if (fiz[0 + n3 * (j + n2 * (i))]>0.0){
            fiz[0 + n3 * (j + n2 * (i))]=0.0;
        }
        if (fez[kmax-2 + n3 * (j + n2 * (i))]<0.0){
            fez[kmax-2 + n3 * (j + n2 * (i))]=0.0;
        }
        if (fiz[kmax-2 + n3 * (j + n2 * (i))]<0.0){
            fiz[kmax-2 + n3 * (j + n2 * (i))]=0.0;
        }
    }
// -----------------------------------------------------------------------------------------------


       for ( I = index_x; I < N; I += stride_x) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
        if (i * j * k == 0 || i >= imax || j >= jmax || k >= kmax) continue;
        ne[k + n3 * (j + n2 * (i))]=ne[k + n3 * (j + n2 * (i))]-dt*(fex[k + n3 * (j + n2 * (i))]-fex[k + n3 * (j + n2 * (i-1))]+fey[k + n3 * (j + n2 * (i))]-fey[k + n3 * (j-1 + n2 * (i))]+fez[k + n3 * (j + n2 * (i))]-fez[k-1 + n3 * (j + n2 * (i))])/h ;
        ni[k + n3 * (j + n2 * (i))]=ni[k + n3 * (j + n2 * (i))]-dt*(fix[k + n3 * (j + n2 * (i))]-fix[k + n3 * (j + n2 * (i-1))]+fiy[k + n3 * (j + n2 * (i))]-fiy[k + n3 * (j-1 + n2 * (i))]+fiz[k + n3 * (j + n2 * (i))]-fiz[k-1 + n3 * (j + n2 * (i))])/h ;
    }





        for ( I = index_x; I < n1 * n2; I += stride_x) {
         j = I % n2;
         i = (I - j) / n2;
        if (i * j == 0 || i >= imax || j >= jmax ) continue;

        ne[0 + n3 * (j + n2 * (i))] = -dt*fez[0 + n3 * (j + n2 * (i))]/h ;
        ni[0 + n3 * (j + n2 * (i))] = -dt*fiz[0 + n3 * (j + n2 * (i))]/h ;

    }

     for ( I = index_x; I < n1 * n3; I += stride_x) {
         k = I % n3;
         i = (I - k) / n3;

        if (i * k == 0 ||  i >= imax || k >= kmax) continue;

        ne[k + n3 * (0 + n2 * (i))] = -dt*fey[k + n3 * (0 + n2 * (i))]/h ;
        ni[k + n3 * (0 + n2 * (i))] = -dt*fiy[k + n3 * (0 + n2 * (i))]/h ;

    }




      for ( I = index_x; I < n2 * n3; I += stride_x) {
         k = I % n3;
         j = (I - k) / n3;
        if (j * k == 0 ||  j >= jmax || k >= kmax) continue;

       ne[k + n3 * (j + n2 * (0))]= -dt*fex[k + n3 * (j + n2 * (0))]/h ;
       ni[k + n3 * (j + n2 * (0))]= -dt*fix[k + n3 * (j + n2 * (0))]/h ;


    }




        // BC on densities

     for ( I = index_x; I < n1 * n3; I += stride_x) {
         k = I % n3;
         i = (I - k) / n3;

        if (i * k == 0 ||  i >= imax || k >= kmax) continue;

        ne[k + n3 * (0 + n2 * (i))] = 0.0 ;
        ni[k + n3 * (0 + n2 * (i))] = 0.0 ;

        ne[k + n3 * (jmax-1 + n2 * (i))] = 0.0 ;
        ni[k + n3 * (jmax-1 + n2 * (i))] = 0.0 ;

    }




      for ( I = index_x; I < n2 * n3; I += stride_x) {
         k = I % n3;
         j = (I - k) / n3;
        if (j * k == 0 ||  j >= jmax || k >= kmax) continue;

       ne[k + n3 * (j + n2 * (0))]= 0.0 ;
       ni[k + n3 * (j + n2 * (0))]= 0.0 ;

       ne[k + n3 * (j + n2 * (imax-1))]= 0.0 ;
       ni[k + n3 * (j + n2 * (imax-1))]= 0.0 ;


    }



    for ( I = index_x; I < n1*n2; I += stride_x) {
         j = I % n2;
         i = (I - j) / n2;
        if (i * j == 0 || i >= imax+1 || j >= jmax+1) continue;
        ne[kmax-1 + n3 * (j + n2 * (i))]=0.0;
        ne[0 + n3 * (j + n2 * (i))]=0.0;
        ni[kmax-1 + n3 * (j + n2 * (i))]=0.0;
        ni[0 + n3 * (j + n2 * (i))]=0.0;

    }




        // calculating the loss
//    sf=0.0;
//      for ( I = index_x; I < N; I += stride_x) {
//        k = I % n3;
//        s1 = (I - k) / n3;
//        j = s1 % n2;
//        i = (s1 - j) / n2;
//        if (i * j * k == 0 || i >= imax || j >= jmax || k >= kmax) continue;
//       sf=sf+ne[k + n3 * (j + n2 * (i))] ;
//   }
//
//       alpha=(si-sf)/sf;
//
//       float tmp;
//   for ( I = index_x; I < N; I += stride_x) {
//        k = I % n3;
//        s1 = (I - k) / n3;
//        j = s1 % n2;
//        i = (s1 - j) / n2;
//       if (i * j * k == 0 || i >= imax-1 || j >= jmax-1 || k >= kmax-1) continue;
//       ne[k + n3 * (j + n2 * (i))]=ne[k + n3 * (j + n2 * (i))]+alpha*ne[k + n3 * (j + n2 * (i))] ;
//       ni[k + n3 * (j + n2 * (i))]=ni[k + n3 * (j + n2 * (i))]+alpha*ne[k + n3 * (j + n2 * (i))] ;
//   }
//   for ( I = index_x; I < N; I += stride_x) {
//       ne[I] = ne_temp[I];
//       ne[I] = ne_temp[I];
//   }
      // if (myTime%100==0.0){
     //  }
//
    *sf_temp = 0;
}

void sum_ne(int imax, int jmax, int kmax, float* mat, float* res) {
    *res = 0;
    int index_x = 0;
    int stride_x = 1;
    int  n1=imax+3, n2 = jmax+3, n3 = kmax+3,i,j,k,myTime,kk,I,N,s1; 
    N=n1*n2*n3;
    for (int I = index_x; I < N; I += stride_x) {
        int k = I % n3;
        int s1 = (I - k) / n3;
        int j = s1 % n2;
        int i = (s1 - j) / n2;
       if (i * j * k == 0 || i >= imax-1 || j >= jmax-1 || k >= kmax-1) continue;
        *res += mat[I];
    }

}

__global__ void sum_ne_cu(int imax, int jmax, int kmax, float* mat, float* res) {
    int index_x = threadIdx.x + blockDim.x * blockIdx.x;
    int stride_x = blockDim.x * gridDim.x;
    int  n1=imax+3, n2 = jmax+3, n3 = kmax+3,i,j,k,myTime,kk,I,N,s1; 
    N=n1*n2*n3;
    for (int I = index_x; I < N; I += stride_x) {
        int k = I % n3;
        int s1 = (I - k) / n3;
        int j = s1 % n2;
        int i = (s1 - j) / n2;
       if (i * j * k == 0 || i >= imax-1 || j >= jmax-1 || k >= kmax-1) continue;
       atomicAdd(res, mat[I]);
    }
}
void update_ne(int imax, int jmax, int kmax, float* ne, float* ni, float *sf, float si) {
    int index_x = 0;
    int stride_x = 1;
    int  n1=imax+3, n2 = jmax+3, n3 = kmax+3,i,j,k,myTime,kk,I,N,s1; 
    N=n1*n2*n3;
    float alpha=(si-sf[0])/(sf[0]);
   for ( int I = index_x; I < N; I += stride_x) {
        int k = I % n3;
        int s1 = (I - k) / n3;
        int j = s1 % n2;
        int i = (s1 - j) / n2;
       if (i * j * k == 0 || i >= imax-1 || j >= jmax-1 || k >= kmax-1) continue;
       ne[k + n3 * (j + n2 * (i))]=ne[k + n3 * (j + n2 * (i))]+alpha*ne[k + n3 * (j + n2 * (i))] ;
       ni[k + n3 * (j + n2 * (i))]=ni[k + n3 * (j + n2 * (i))]+alpha*ne[k + n3 * (j + n2 * (i))] ;
   }

}__global__ void update_ne_cu(int imax, int jmax, int kmax, float* ne, float* ni, float *sf, float si) {
    int index_x = threadIdx.x + blockDim.x * blockIdx.x;
    int stride_x = blockDim.x * gridDim.x;
    int  n1=imax+3, n2 = jmax+3, n3 = kmax+3,i,j,k,myTime,kk,I,N,s1; 
    N=n1*n2*n3;
    float alpha=(si-sf[0])/(sf[0]);
   for ( int I = index_x; I < N; I += stride_x) {
        int k = I % n3;
        int s1 = (I - k) / n3;
        int j = s1 % n2;
        int i = (s1 - j) / n2;
       if (i * j * k == 0 || i >= imax-1 || j >= jmax-1 || k >= kmax-1) continue;
       ne[k + n3 * (j + n2 * (i))]=ne[k + n3 * (j + n2 * (i))]+alpha*ne[k + n3 * (j + n2 * (i))] ;
       ni[k + n3 * (j + n2 * (i))]=ni[k + n3 * (j + n2 * (i))]+alpha*ne[k + n3 * (j + n2 * (i))] ;
   }

}

int main()
{
    int imax = 32, jmax = 32, kmax = 64,i,j,k;
    int n1 = imax+3, n2 = jmax+3, n3 = kmax+3;
    float qi=1.6E-19,qe=-1.6E-19, kr = 0,ki = 0,si = 0,sf = 0,alpha = 0, q=1.6E-19,pie=3.14159,Ta,w,eps0,Te,Ti,B,Kb,me,mi,nue,nui,denominator_e,denominator_i,nn,dt,h,wce,wci,mue,mui,dife,difi;
    int tmax = 200;
    float *ne;
    float *ni;
    float *ne_temp;
    float *ni_temp;
    float *difxne;
    float *difyne;
    float *difxni;
    float *difyni;
    float *difxyne;
    float *difxyni;
    float *Exy;
    float *fexy;
    float *fixy;
    float *g;
    float *g_temp;
    float *R;
    float *Ex;
    float *Ey;
    float *fex;
    float *fey;
    float *fix;
    float *fiy;
    float *V;
    float *L;
    float *difzne;
    float *difzni;
    float *Ez;
    float *fez;
    float *fiz;
    hipMallocManaged(&(ne ), n1 * n2 * n3 * sizeof(float));
    hipMallocManaged(&(ni ), n1 * n2 * n3 * sizeof(float));
    hipMallocManaged(&(ne_temp ), n1 * n2 * n3 * sizeof(float));
    hipMallocManaged(&(ni_temp ), n1 * n2 * n3 * sizeof(float));
    hipMallocManaged(&(difxne ), n1 * n2 * n3 * sizeof(float));
    hipMallocManaged(&(difyne ), n1 * n2 * n3 * sizeof(float));
    hipMallocManaged(&(difxni ), n1 * n2 * n3 * sizeof(float));
    hipMallocManaged(&(difyni ), n1 * n2 * n3 * sizeof(float));
    hipMallocManaged(&(difxyne ), n1 * n2 * n3 * sizeof(float));
    hipMallocManaged(&(difxyni ), n1 * n2 * n3 * sizeof(float));
    hipMallocManaged(&(Exy ), n1 * n2 * n3 * sizeof(float));
    hipMallocManaged(&(fexy ), n1 * n2 * n3 * sizeof(float));
    hipMallocManaged(&(fixy ), n1 * n2 * n3 * sizeof(float));
    hipMallocManaged(&(g ), n1 * n2 * n3 * sizeof(float));
    hipMallocManaged(&(g_temp ), n1 * n2 * n3 * sizeof(float));
    hipMallocManaged(&(R ), n1 * n2 * n3 * sizeof(float));
    hipMallocManaged(&(Ex ), n1 * n2 * n3 * sizeof(float));
    hipMallocManaged(&(Ey ), n1 * n2 * n3 * sizeof(float));
    hipMallocManaged(&(fex ), n1 * n2 * n3 * sizeof(float));
    hipMallocManaged(&(fey ), n1 * n2 * n3 * sizeof(float));
    hipMallocManaged(&(fix ), n1 * n2 * n3 * sizeof(float));
    hipMallocManaged(&(fiy ), n1 * n2 * n3 * sizeof(float));
    hipMallocManaged(&(V ), n1 * n2 * n3 * sizeof(float));
    hipMallocManaged(&(L ), n1 * n2 * n3 * sizeof(float));
    hipMallocManaged(&(difzne ), n1 * n2 * n3 * sizeof(float));
    hipMallocManaged(&(difzni ), n1 * n2 * n3 * sizeof(float));
    hipMallocManaged(&(Ez ), n1 * n2 * n3 * sizeof(float));
    hipMallocManaged(&(fez ), n1 * n2 * n3 * sizeof(float));
    hipMallocManaged(&(fiz ), n1 * n2 * n3 * sizeof(float));

    Kb    = 1.38E-23;
    B     = 0.0;
    Te    = 2.5*11604.5;
    Ti    = 0.025*11604.5;
    me    = 9.109E-31;
    mi    = 6.633E-26;
    ki    = 0.0;
    dt    = 1.0E-12;
    h     = 1.0E-3;
    eps0  = 8.854E-12;
    si    = 0.0;
    sf    =0.0;

    for ( i=0; i<imax+3;i++){
        for ( j=0; j<jmax+3; j++){
            for ( k=0; k<kmax+3;k++){
                ne[k + n3 * (j + n2 * (i))] = 1e-9;
                ni[k + n3 * (j + n2 * (i))] = 1e-9;
                difxne[k + n3 * (j + n2 * (i))] = 1e-9;
                difyne[k + n3 * (j + n2 * (i))] = 1e-9;
                difxni[k + n3 * (j + n2 * (i))] = 1e-9;
                difyni[k + n3 * (j + n2 * (i))] = 1e-9;
                difxyne[k + n3 * (j + n2 * (i))] = 1e-9;
                difxyni[k + n3 * (j + n2 * (i))] = 1e-9;
                Exy[k + n3 * (j + n2 * (i))] = 1e-9;
                fexy[k + n3 * (j + n2 * (i))] = 1e-9;
                fixy[k + n3 * (j + n2 * (i))] = 1e-9;
                g[k + n3 * (j + n2 * (i))] = 1e-9;
                R[k + n3 * (j + n2 * (i))] = 1e-9;
                Ex[k + n3 * (j + n2 * (i))] = 1e-9;
                Ey[k + n3 * (j + n2 * (i))] = 1e-9;
                fex[k + n3 * (j + n2 * (i))] = 1e-9;
                fey[k + n3 * (j + n2 * (i))] = 1e-9;
                fix[k + n3 * (j + n2 * (i))] = 1e-9;
                fiy[k + n3 * (j + n2 * (i))] = 1e-9;
                V[k + n3 * (j + n2 * (i))] = 1e-9;
                L[k + n3 * (j + n2 * (i))] = 1e-9;
                difzne[k + n3 * (j + n2 * (i))] = 1e-9;
                difzni[k + n3 * (j + n2 * (i))] = 1e-9;
                Ez[k + n3 * (j + n2 * (i))] = 1e-9;
                fez[k + n3 * (j + n2 * (i))] = 1e-9;
                fiz[k + n3 * (j + n2 * (i))] = 1e-9;
             }
        }
    }


    nn=1.33/(Kb*Ti); //neutral density=p/(Kb.T)
    nue=nn*1.1E-19*sqrt(Kb*Te/me); // electron collision frequency= neutral density * sigma_e*Vth_e
    nui=nn*4.4E-19*sqrt(Kb*Ti/mi);
    wce=q*B/me;
    wci=q*B/mi;
    mue=q/(me*nue);
    mui=q/(mi*nui);
    dife=Kb*Te/(me*nue);
    difi=Kb*Ti/(mi*nui);
    ki=0.00002/(nn*dt);
    denominator_e= (1+wce*wce/(nue*nue));
    denominator_i= (1+wci*wci/(nui*nui));
    // Ta and W are just some constants needed for the iterative method that we have used to solve Poisson eq.
    Ta=acos((cos(pie/imax)+cos(pie/jmax)+cos(pie/kmax))/3.0);// needs to be float checked
    w=2.0/(1.0+sin(Ta));
// -----------------------------------------------------------------------------------------------
    //Density initialization
    // To add multiple Gaussian sources, just simply use the density_initialization function at the (x,y) points that you want
    int x_position = 15, y_position = 15, z_position = 15;
    for ( i=1; i<imax-1;i++){
        for ( j=1; j<jmax-1;j++){
            for ( k=1; k<kmax-1;k++){
            ne[k + n3 * (j + n2 * (i))]= 2.0E13;/*
                1.0E14+1.0E14*exp(-(pow((i-x_position),2)+
                pow((j-y_position),2)+pow((k-z_position),2))/100.0);*/
            ni[k + n3 * (j + n2 * (i))]=2.0E13;/*
                1.0E14+1.0E14*exp(-(pow((i-x_position),2)+
                pow((j-y_position),2)+pow((k-z_position),2))/100.0);*/
            }
        }
    }



    for ( k=1; k<kmax+1; k++) {
        for ( j=1; j<jmax+1; j++) {
            for ( i=1; i<imax+1;i++) {
            si=si+ne[k + n3 * (j + n2 * (i))] ;
            }
        }
    }

    int myTime,kk,I,N,s1; 
    N=n1*n2*n3;
    int iterations = 40;

    float* sf_temp;
    hipMallocManaged(&sf_temp, sizeof(sf_temp));
    double begin = clock();
    for ( myTime=1; myTime<tmax; myTime++){  // This for loop takes care of myTime evolution
        printf("%d\n", myTime);
//################################################################################################################
        //UNCOMMENT THE FOLLOWING TO RUN THE CODE ON CPU
//################################################################################################################
        //before_poisson(imax, jmax, kmax, tmax, ne, ni, difxne, difyne, difxni, difyni, difxyne, difxyni, Exy, fexy, fixy, g, g_temp, R, Ex, Ey, fex, fey, fix, fiy, V, L, difzne, difzni, Ez, fez, fiz, qi, qe, kr, ki, si, sf, alpha, q, pie,Ta ,w ,eps0 , Te, Ti, B, Kb, me, mi, nue, nui, denominator_e, denominator_i, nn, dt, h, wce, wci, mue, mui, dife, difi);

        //poisson_solve(imax, jmax, kmax, n1, n2, n3, N, iterations, V, g, R, w, h);

        //after_poisson(imax, jmax, kmax, tmax, ne, ni, ne_temp, ni_temp, difxne, difyne, difxni, difyni, difxyne, difxyni, Exy, fexy, fixy, g, g_temp, R, Ex, Ey, fex, fey, fix, fiy, V, L, difzne, difzni, Ez, fez, fiz, qi, qe, kr, ki, si, sf, alpha, q, pie,Ta ,w ,eps0 , Te, Ti, B, Kb, me, mi, nue, nui, denominator_e, denominator_i, nn, dt, h, wce, wci, mue, mui, dife, difi);
        //sum_ne(imax, jmax, kmax, ne, sf_temp);
        //printf("sf = %f\n", *sf_temp);
        //update_ne(imax, jmax, kmax, ne, ni, sf_temp, si);


//################################################################################################################
        //UNCOMMENT THE FOLLOWING TO RUN THE CODE ON GPU
//################################################################################################################
        before_poisson_cu<<<256, N/256>>>(imax, jmax, kmax, tmax, ne, ni, difxne, difyne, difxni, difyni, difxyne, difxyni, Exy, fexy, fixy, g, g_temp, R, Ex, Ey, fex, fey, fix, fiy, V, L, difzne, difzni, Ez, fez, fiz, qi, qe, kr, ki, si, sf, alpha, q, pie,Ta ,w ,eps0 , Te, Ti, B, Kb, me, mi, nue, nui, denominator_e, denominator_i, nn, dt, h, wce, wci, mue, mui, dife, difi);
        for (kk=0; kk<iterations; kk++) {
            poisson_solve_1it_cu<<<256, N/256>>>(imax, jmax, kmax, n1, n2, n3, N, iterations, V, g, R, w, h, 1);
            poisson_solve_1it_cu<<<256, N/256>>>(imax, jmax, kmax, n1, n2, n3, N, iterations, V, g, R, w, h, 0);
        }
        after_poisson_cu<<<256, N/256>>>(imax, jmax, kmax, tmax, ne, ni, ne_temp, ni_temp, difxne, difyne, difxni, difyni, difxyne, difxyni, Exy, fexy, fixy, g, g_temp, R, Ex, Ey, fex, fey, fix, fiy, V, L, difzne, difzni, Ez, fez, fiz, qi, qe, kr, ki, si, sf, alpha, q, pie,Ta ,w ,eps0 , Te, Ti, B, Kb, me, mi, nue, nui, denominator_e, denominator_i, nn, dt, h, wce, wci, mue, mui, dife, difi, sf_temp); 
        sum_ne_cu<<<256, N/256>>>(imax, jmax, kmax, ne, sf_temp);
        update_ne_cu<<<256, N/256>>>(imax, jmax, kmax, ne, ni, sf_temp, si);
        //hipDeviceSynchronize();
        //printf("sf = %f\n", *sf_temp);
        //printf("si = %f\n", si);
     }

    hipDeviceSynchronize();
    double time_spent1 = (clock() - begin) / CLOCKS_PER_SEC;
    printf("Time spent without parallelization: %f\n", time_spent1);

    printf("%f\n", V[5 + (kmax+3) * (5 + (jmax+3) * 5)]);
}
