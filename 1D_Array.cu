#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define my_index 3

typedef struct {
    float **A;
    float **B;
} Mats;

typedef struct {
    float **indices;
    float **values;
    int *sizes;
} sparseMat;

typedef struct {
    sparseMat *A;
    sparseMat *B;
} sparseMats;

typedef struct {
    float *array;
    size_t used;
    size_t size;
} Array;
float rel_error(int n, float *a, float *b) {
    float max_diff = -100000;
    float tmp, tmp1, tmp2, sum;
    for (int i=0; i < n; i++) {
        tmp = a[i] - b[i];
        if (tmp < 0) tmp = -tmp;

        tmp1 = a[i];
        tmp2 = b[i];
        if (tmp1 < 0) tmp1 = -tmp1;
        if (tmp2 < 0) tmp2 = -tmp2;
        sum = tmp1 + tmp1;
        if (sum < 1e-8) sum=1e-8;
        if (tmp / sum > max_diff) max_diff=tmp/sum;
    }
    return max_diff;
}
float sum(int n, float *a) {
    float s = 0;
    for (int i=0; i<n; i++) {
        s += a[i];
    }
    return s;
}
float sum2(int n, float **a) {
    float s = 0;
    for (int i=0; i<n; i++) {
        for (int j=0; j<n; j++) {
            s += a[i][j];
        }
    }
    return s;
}
void initArray(Array *a, size_t initialSize) {
    a->array = (float *)malloc(initialSize * sizeof(float));
    //hipMallocManaged(&(a->array), initialSize * sizeof(float));
    a->used = 0;
    a->size = initialSize;
}


void insertArray(Array *a, float element) {
    // a->used is the number of used entries, 
    //because a->array[a->used++] updates a->used 
    //only *after* the array has been accessed.
    // Therefore a->used can go up to a->size 
    if (a->used == a->size) {
        printf("reallocating\n");
        a->size *= 2;
        a->array = (float *)realloc(a->array, a->size * sizeof(float));
    }
    a->array[a->used++] = element;
}
void freeArray(Array *a) {
    free(a->array);
    a->array = NULL;
    a->used = a->size = 0;
}

void square_matrix_to_sparse(int N, float **mat, Array **indices, Array **values) {
    for (int i=0; i<N;i++){
        for(int j=0;j<N;j++) {
            if (mat[i][j]) {
                insertArray(indices[i], (float) j);
                insertArray(values[i], mat[i][j]);
            }
        }
    }
}

void mat_dot_vec(int N, float **I_kernel, float *V, float *R) {
    for (int i=0; i < N; i++) {
        R[i] = 0;
        for (int j=0; j < N; j++) {
            R[i] += I_kernel[i][j]*V[j];
        }
    }
}

void mat_dot_mat(int N, float **I_kernel, float **V, float **R) {
    for (int i=0; i < N; i++) {
        for (int j=0; j < N; j++) {
            R[i][j] = 0;
            for (int k=0; k < N; k++) R[i][j] += I_kernel[i][k]*V[k][j];
        }
    }
}

void mat_add_mat(int N, float **I_kernel, float **V, float **R) {
    for (int i=0; i < N; i++) {
        for (int j=0; j < N; j++) {
            R[i][j] = I_kernel[i][j]+V[i][j];
        }
    }
}

void vec_min_vec(int N, float *I_kernel, float *V, float *R) {
    for (int i=0; i < N; i++) {
        R[i] = I_kernel[i]-V[i];
    }
}

void array_dot_vec(int N, float **indices, float **values, int *sizes, float *V, float *R) {
    for (int i=0; i < N; i+=1) {
        R[i] = 0;
        for (int j=0; j < sizes[i]; j+=1) {
            R[i] += values[i][j] * V[(int)indices[i][j]];
        }
    }
}

float** create_kernel(int imax, int jmax, int kmax, float w) {
    int n1 = imax+3;
    int n2 = jmax+3;
    int n3 = kmax+3;
    int N = n1*n2*n3;
    float **my_mat;
    //hipMallocManaged(&my_mat, N*sizeof(my_mat));
    my_mat = (float**) malloc(N*sizeof(my_mat));
    for (int i=0; i<N; i++) {
        //hipMallocManaged(&my_mat[i], N*sizeof(my_mat[i]));
        my_mat[i] = (float*) malloc(N*sizeof(my_mat[i]));
    }
    for (int I=0; I<N; I++) {
        for (int J=0; J<N; J++) my_mat[I][J] = 0;
        int k = I % n3;
        int s1 = (I - k) / n3;
        int j = s1 % n2;
        int i = (s1 - j) / n2;
        if (I % 1000 == 0) printf("%4d / %d\n", I, N);
        if (i >= 1 && i < imax-1 
                && j >= 1 && j < jmax-1 
                && k >= 1 && k < kmax-1) {
            for (int J=0; J<N; J++)  {
                my_mat[I][J] = w / 6. *
                (my_mat[I-n2*n3][J] +
                my_mat[I-n2][J] +
                my_mat[I-1][J]);
            }
            my_mat[I][I+1] += w / 6.;
            my_mat[I][I+n3] += w / 6.;
            my_mat[I][I+n2*n3] += w / 6.;
            my_mat[I][I] += 1 - w;
        }
        else
            my_mat[I][I] = 1;

    }
    printf("w = %f\n", w);
    printf("sum of kernel: %f\n", sum2(N, my_mat));
    return my_mat;
}



sparseMat* dense_to_sparse(int N, float** my_mat) {
    Array **indices;
    Array **values;
    indices = (Array **) malloc(N * sizeof(indices));
    values  = (Array **) malloc(N * sizeof(values));
    for (int i=0; i < N; i++) {
        indices[i] = (Array *) malloc(sizeof(indices[i]));
        values[i] = (Array *) malloc(sizeof(values[i]));
        initArray(indices[i], N);
        initArray(values[i], N);
    }
    //Converting normal matrix to sparse
    //the result would be three arrays
    // indices, values and sizes
    square_matrix_to_sparse(N, my_mat, indices, values);
    float **indices_t;
    float **values_t;
    int *sizes_t;
    //hipMallocManaged(&indices_t, N*sizeof(indices_t));
    //hipMallocManaged(&values_t, N*sizeof(values_t));
    //hipMallocManaged(&size_t, N*sizeof(size_t));
    indices_t = (float**) malloc(N*sizeof(indices_t));
    values_t = (float**) malloc(N*sizeof(values_t));
    sizes_t = (int*) malloc(N*sizeof(values_t));
    for (int i=0; i<N; i++) {
        sizes_t[i] = indices[i]->used;
        //hipMallocManaged(&indices_t[i], size_t[i]*sizeof(indices_t[i]));
        //hipMallocManaged(&values_t[i], size_t[i]*sizeof(values_t[i]));
        indices_t[i] = (float*) malloc(sizes_t[i]*sizeof(indices_t[i]));
        values_t[i] = (float*) malloc(sizes_t[i]*sizeof(values_t[i]));
        for (int j=0; j<sizes_t[i]; j++) {
            indices_t[i][j] = indices[i]->array[j];
            values_t[i][j] = values[i]->array[j];
        }
    }
    sparseMat *sp = (sparseMat *) malloc(sizeof(sparseMat));
    sp->indices=indices_t;
    sp->values=values_t;
    sp->sizes=sizes_t;
    return sp;
}

sparseMats* createAB(int N, float **kernel, int iterations, float **A, float **B, float **holder1) {
    for (int i=0; i<N; i++) {
        for (int j=0; j<N; j++) {
            A[i][j] = kernel[i][j];
            B[i][j] = 0;
            if (i==j) B[i][j] = 1;
        }
    }
    for (int kk=0; kk < iterations-1; kk++) {
        printf("iteration %d\n", kk);
        mat_add_mat(N, B, A, B);
        mat_dot_mat(N, kernel, A, holder1);
        float **tmp = holder1;
        holder1 = A;
        A = tmp;
    }
    sparseMats *AB = (sparseMats *) malloc(sizeof(sparseMats));
    AB->A = dense_to_sparse(N, A);
    AB->B = dense_to_sparse(N, B);
    printf("sum of A: %f", sum2(N, A));
    return AB;
}

void poisson_solve3(int imax, int jmax, int kmax, 
    int n1, int n2, int n3, int N, int iterations, 
    float* V, float* g, float* g_temp, float *R, float w,
    float h, sparseMat* kernel, float** kernel2) {
        for (int I=0; I<N; I++) g_temp[I] = w*h*h*g[I]/6.;
        for (int I=0; I<N; I++) {
            int k = I % n3;
            int s1 = (I - k) / n3;
            int j = s1 % n2;
            int i = (s1 - j) / n2;
            if (i >= 1 && i < imax-1 
                    && j >= 1 && j < jmax-1 
                    && k >= 1 && k < kmax-1) {
                g_temp[I] += w/6.*(g_temp[I-1]+g_temp[I-n3]+g_temp[I-n3*n2]);
            }
            else {
                g_temp[I] = 0;
            }
        }

        array_dot_vec(N, AB->A->indices, AB->A->values, AB->A->sizes, V, R);
        array_dot_vec(N, AB->B->indices, AB->B->values, AB->B->sizes, g_temp, holder);
        vec_min_vec(N, R, holder, V);
        //printf("\n");
    }

void poisson_solve2(int imax, int jmax, int kmax, 
    int n1, int n2, int n3, int N, int iterations, 
    float* V, float* g, float* g_temp, float *R, float w,
    float h, sparseMat* kernel, float** kernel2) {
        for (int I=0; I<N; I++) g_temp[I] = w*h*h*g[I]/6.;
        for (int I=0; I<N; I++) {
            int k = I % n3;
            int s1 = (I - k) / n3;
            int j = s1 % n2;
            int i = (s1 - j) / n2;
            if (i >= 1 && i < imax-1 
                    && j >= 1 && j < jmax-1 
                    && k >= 1 && k < kmax-1) {
                g_temp[I] += w/6.*(g_temp[I-1]+g_temp[I-n3]+g_temp[I-n3*n2]);
            }
            else {
                g_temp[I] = 0;
            }
        }
        for (int kk=0; kk<iterations; kk++) {
            array_dot_vec(N, kernel->indices, kernel->values, kernel->sizes, V, R);
            //mat_dot_vec(N, kernel2, V, R);
            for (int i=0; i<N; i++) V[i] = R[i] - g_temp[i];
            //printf("%f, ", sum(N, V));
        }
        //printf("\n");
    }

void poisson_solve(int imax, int jmax, int kmax, int n1, int n2, int n3, int N, int iterations, float* V, float* g, float *R, float w, float h) {
      for (int kk=0; kk<iterations; kk++) {
        for (int I = 0; I < N; I ++) {
             int k = I % n3;
             int s1 = (I - k) / n3;
             int j = s1 % n2;
             int i = (s1 - j) / n2;
            if (i * j * k == 0 || i >= imax-1 || j >= jmax-1 || k >= kmax-1) continue;
            R[k + n3 * (j + n2 * (i))]=
                (V[k + n3 * (j + n2 * (i+1))]+
                     V[k + n3 * (j + n2 * (i-1))]+
                     V[k + n3 * (j+1 + n2 * (i))]+
                     V[k + n3 * (j-1 + n2 * (i))]+
                     V[k+1 + n3 * (j + n2 * (i))]+
                     V[k-1 + n3 * (j + n2 * (i))]
                 ) / 6.0 - V[k + n3 * (j + n2 * (i))]- (h*h)*g[k + n3 * (j + n2 * (i))]/6.0;
            V[k + n3 * (j + n2 * (i))] += w*R[k + n3 * (j + n2 * (i))];
        }
    }
}

void mardas(int imax, int jmax, int kmax, int tmax, float *ne, float *ni, float *difxne, float *difyne, float *difxni,
             float *difyni, float *difxyne, float *difxyni, float *Exy, float *fexy, float *fixy, float *g, float* g_temp, float *R,
              float *Ex, float *Ey, float *fex, float *fey, float *fix, float *fiy, float *V, float *L, float *difzne,
               float *difzni, float *Ez, float *fez, float *fiz, float qi, float qe, float kr, float ki, float si,
                float sf, float alpha, float q, float pie, float Ta , float w , float eps0 , float Te, float Ti,
                 float B, float Kb, float me, float mi, float nue, float nui, float denominator_e, float denominator_i,
                  float nn, float dt, float h, float wce, float wci, float mue, float mui, float dife, float difi) {
    int  n1=imax+3, n2 = jmax+3, n3 = kmax+3,i,j,k,fuckingCount,myTime,kk,I,N,s1;

    N=n1*n2*n3;
float** kernel = create_kernel(imax, jmax, kmax, w);
sparseMat* sp = dense_to_sparse(N, kernel);
float** A;
float** BB;
float** holder1;
float* holder2;
A = (float**) malloc(N*sizeof(A));
BB = (float**) malloc(N*sizeof(BB));
holder1 = (float**) malloc(N*sizeof(holder1));
holder2 = (float*) malloc(N*sizeof(holder2));
for (int i=0; i<N; i++) {
    //hipMallocManaged(&my_mat[i], N*sizeof(my_mat[i]));
    A[i] = (float*) malloc(N*sizeof(A[i]));
    BB[i] = (float*) malloc(N*sizeof(BB[i]));
    holder1[i] = (float*) malloc(N*sizeof(holder1[i]));
}
int iterations = 5;
//sparseMats* AB = createAB(N, kernel, iterations, A, BB, holder1);

for ( myTime=1; myTime<tmax; myTime++){  // This for loop takes care of myTime evolution
     fuckingCount = 0;


    //    printf("time %d_%d V: %f\n", myTime, fuckingCount, V[3 + n3 * (3 + n2 * (3))]);
    //    printf("time %d_%d g: %f\n", myTime, fuckingCount, g[3 + n3 * (3 + n2 * (3))]);
    //    printf("time %d_%d ne: %f\n", myTime, fuckingCount, ne[3 + n3 * (3 + n2 * (3))]);
// -----------------------------------------------------------------------------------------------


        printf("%d \n", myTime);

      // Solving P//oisson's eq. to get the voltage everywhere
  // Here we calculate the right hand side of the Poisson equation



    for ( I = 0; I < N; I ++) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
        if (i * j * k == 0 || i >= imax-1 || j >= jmax-1 || k >= kmax-1) continue;
        g[k + n3 * (j + n2 * (i))]=-(ne[k + n3 * (j + n2 * (i))]*qe+ni[k + n3 * (j + n2 * (i))]*qi)/eps0;
    }



/////!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
//// has problem at the edges
/////!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
// solving Poisson eq. using  successive over-relaxation method

    //poisson_solve(imax, jmax, kmax, n1, n2, n3, N, iterations, V, g, R, w, h);
    poisson_solve2(imax, jmax, kmax, n1, n2, n3, N, iterations, V, g, g_temp, R, w, h, sp, kernel);
    //poisson_solve3(imax, jmax, kmax, n1, n2, n3, N, iterations, V, g, g_temp, R, w, h, AB, holder2);



        ///printf("time %d_%d V: %f\n", myTime, fuckingCount, V[3 + n3 * (3 + n2 * (3))]);
        ///printf("time %d_%d g: %f\n", myTime, fuckingCount, g[3 + n3 * (3 + n2 * (3))]);
        ///printf("time %d_%d ne: %f\n", myTime, fuckingCount, ne[3 + n3 * (3 + n2 * (3))]);

        ///printf("qi = %f\n", qi);
        ///printf("qe = %f\n", qe);
        ///printf("kr = %f\n", kr);
        ///printf("ki = %f\n", ki);
        ///printf("si = %f\n", si);
        ///printf("sf = %f\n", sf);
        ///printf("alpha = %f\n", alpha);
        ///printf("q = %f\n", q);
        ///printf("pie = %f\n", pie);
        ///printf("Ta = %f\n", Ta);
        ///printf("w = %f\n", w);
        ///printf("eps0 = %f\n", eps0);
        ///printf("Te = %f\n", Te);
        ///printf("Ti = %f\n", Ti);
        ///printf("B = %f\n", B);
        ///printf("Kb = %f\n", Kb);
        ///printf("me = %f\n", me);
        ///printf("mi = %f\n", mi);
        ///printf("nue = %f\n", nue);
        ///printf("nui = %f\n", nui);
        ///printf("denominator_e = %f\n", denominator_e);
        ///printf("denominator_i = %f\n", denominator_i);
        ///printf("nn = %f\n", nn);
        ///printf("dt = %f\n", dt);
        ///printf("h = %f\n", h);
        ///printf("wce = %f\n", wce);
        ///printf("wci = %f\n", wci);
        ///printf("mue = %f\n", mue);
        ///printf("mui = %f\n", mui);
        ///printf("dife = %f\n", dife);
        ///printf("difi = %f\n", difi);
// -----------------------------------------------------------------------------------------------
      // Now Calculating electric field and density gradient which are calculated through function electric_field_elements

  for ( I = 0; I < N; I ++) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
        if (i >= imax-1 || j >= jmax || k >= kmax) continue;
        Ex[k + n3 * (j + n2 * (i))]= (V[k + n3 * (j + n2 * (i))]-V[k + n3 * (j + n2 * (i+1))])/h;
        difxne[k + n3 * (j + n2 * (i))]=(ne[k + n3 * (j + n2 * (i+1))]-ne[k + n3 * (j + n2 * (i))])/h;
        difxni[k + n3 * (j + n2 * (i))]=(ni[k + n3 * (j + n2 * (i+1))]-ni[k + n3 * (j + n2 * (i))])/h;
        }


    for ( I = 0; I < N; I ++) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
        if (i >= imax || j >= jmax-1 || k >= kmax) continue;
        Ey[k + n3 * (j + n2 * (i))]= (V[k + n3 * (j + n2 * (i))]-V[k + n3 * (j+1 + n2 * (i))])/h;
        difyne[k + n3 * (j + n2 * (i))]=(ne[k + n3 * (j+1 + n2 * (i))]-ne[k + n3 * (j + n2 * (i))])/h;
        difyni[k + n3 * (j + n2 * (i))]=(ni[k + n3 * (j+1 + n2 * (i))]-ni[k + n3 * (j + n2 * (i))])/h;
        }


    for ( I = 0; I < N; I ++) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
        if (i >= imax || j >= jmax || k >= kmax-1) continue;
       Ez[k + n3 * (j + n2 * (i))]= (V[k + n3 * (j + n2 * (i))]-V[k+1 + n3 * (j + n2 * (i))])/h;
       difzne[k + n3 * (j + n2 * (i))]=(ne[k+1 + n3 * (j + n2 * (i))]-ne[k + n3 * (j + n2 * (i))])/h;
       difzni[k + n3 * (j + n2 * (i))]=(ni[k+1 + n3 * (j + n2 * (i))]-ni[k + n3 * (j + n2 * (i))])/h;
     }

// -----------------------------------------------------------------------------------------------
       /* Since I am using mid points for Calculating electric field and density gradient,
        to calculate Ex at any point that I don't have it directly, the average over
        the neighboring points is used instead. these average variables are, exy, fexy, fixy, ...*/
        // Calculating the average values of Ex and gradiant_x
   for ( I = 0; I < N; I ++) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;

        Exy[k + n3 * (j + n2 * (i))]= 0.0 ;
        difxyne[k + n3 * (j + n2 * (i))]=0.0;
        difxyni[k + n3 * (j + n2 * (i))]=0.0;
    }

    for ( I = 0; I < N; I ++) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
        if (i * k == 0 ||  i >= imax-1 || j >= jmax-1 || k >= kmax-1) continue;
        Exy[k + n3 * (j + n2 * (i))]= 0.25*(Ex[k + n3 * (j + n2 * (i))]+Ex[k + n3 * (j+1 + n2 * (i))]+Ex[k + n3 * (j + n2 * (i-1))]+Ex[k + n3 * (j+1 + n2 * (i-1))]) ;
        difxyne[k + n3 * (j + n2 * (i))]=0.25*(difxne[k + n3 * (j + n2 * (i))]+difxne[k + n3 * (j+1 + n2 * (i))]+difxne[k + n3 * (j + n2 * (i-1))]+difxne[k + n3 * (j+1 + n2 * (i-1))]);
        difxyni[k + n3 * (j + n2 * (i))]=0.25*(difxni[k + n3 * (j + n2 * (i))]+difxni[k + n3 * (j+1 + n2 * (i))]+difxni[k + n3 * (j + n2 * (i-1))]+difxni[k + n3 * (j+1 + n2 * (i-1))]);
    }


// -----------------------------------------------------------------------------------------------
        // Here we calculate the fluxes in y direction

       for ( I = 0; I < N; I ++) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
        if (i * k == 0 ||  i >= imax-1 || j >= jmax-1 || k >= kmax-1) continue;
        fey[k + n3 * (j + n2 * (i))]= (-0.5*(ne[k + n3 * (j+1 + n2 * (i))]+ne[k + n3 * (j + n2 * (i))])*mue*Ey[k + n3 * (j + n2 * (i))]-dife*difyne[k + n3 * (j + n2 * (i))]
        -wce*q*0.5*(ne[k + n3 * (j+1 + n2 * (i))]+ne[k + n3 * (j + n2 * (i))])*Exy[k + n3 * (j + n2 * (i))]/(me*nue*nue)-wce*dife*difxyne[k + n3 * (j + n2 * (i))]/nue)/denominator_e;
        fiy[k + n3 * (j + n2 * (i))]= (0.5*(ni[k + n3 * (j+1 + n2 * (i))]+ni[k + n3 * (j + n2 * (i))])*mui*Ey[k + n3 * (j + n2 * (i))]-difi*difyni[k + n3 * (j + n2 * (i))]
        -wci*q*0.5*(ni[k + n3 * (j+1 + n2 * (i))]+ni[k + n3 * (j + n2 * (i))])*Exy[k + n3 * (j + n2 * (i))]/(mi*nui*nui)+wci*difi*difxyni[k + n3 * (j + n2 * (i))]/nui)/denominator_i;
    }


    for ( I =0; I < n1 * n3; I ++) {
         k = I % n3;
         i = (I - k) / n3;

        if (i * k == 0 ||  i >= imax-1 || k >= kmax-1) continue;

        if (fey[k + n3 * (0 + n2 * (i))] > 0.0){
                fey[k + n3 * (0 + n2 * (i))] = 0.0;
                }

        if (fiy[k + n3 * (0 + n2 * (i))] > 0.0){
                fiy[k + n3 * (0 + n2 * (i))] = 0.0;
                }

        if (fey[k + n3 * (jmax-2 + n2 * (i))] < 0.0){
                fey[k + n3 * (jmax-2 + n2 * (i))] = 0.0;
                }

        if (fiy[k + n3 * (jmax-2 + n2 * (i))] < 0.0){
                fiy[k + n3 * (jmax-2 + n2 * (i))] = 0.0;
                }

    }

// -----------------------------------------------------------------------------------------------
        // Calculating the average Exy and difxy to be used in x direction fluxes
       // Calculating the average values of Ey and gradiant_y


      for ( I = 0; I < N; I ++) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;

        Exy[k + n3 * (j + n2 * (i))]= 0.0 ;
        difxyne[k + n3 * (j + n2 * (i))]=0.0;
        difxyni[k + n3 * (j + n2 * (i))]=0.0;
    }


      for ( I = 0; I < N; I ++) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
        if (j * k == 0 || i >= imax-1 || j >= jmax-1 || k >= kmax-1) continue;
        Exy[k + n3 * (j + n2 * (i))]= 0.25*(Ey[k + n3 * (j + n2 * (i))]+Ey[k + n3 * (j-1 + n2 * (i))]+Ey[k + n3 * (j + n2 * (i+1))]+Ey[k + n3 * (j-1 + n2 * (i+1))]);
        difxyne[k + n3 * (j + n2 * (i))]= 0.25*(difyne[k + n3 * (j + n2 * (i))]+difyne[k + n3 * (j-1 + n2 * (i))]+difyne[k + n3 * (j + n2 * (i+1))]+difyne[k + n3 * (j-1 + n2 * (i+1))]);
        difxyni[k + n3 * (j + n2 * (i))]= 0.25*(difyni[k + n3 * (j + n2 * (i))]+difyni[k + n3 * (j-1 + n2 * (i))]+difyni[k + n3 * (j + n2 * (i+1))]+difyni[k + n3 * (j-1 + n2 * (i+1))]);

    }

// -----------------------------------------------------------------------------------------------
        // Now ready to calculate the fluxes in x direction

    for ( I = 0; I < N; I ++) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
        if (j * k == 0 || i >= imax-1 || j >= jmax-1 || k >= kmax-1) continue;
        fex[k + n3 * (j + n2 * (i))]=(-0.5*(ne[k + n3 * (j + n2 * (i))]+ne[k + n3 * (j + n2 * (i+1))])*mue*Ex[k + n3 * (j + n2 * (i))]-dife*difxne[k + n3 * (j + n2 * (i))]
        +wce*dife*difxyne[k + n3 * (j + n2 * (i))]/nue+wce*q*0.5*(ne[k + n3 * (j + n2 * (i))]+ne[k + n3 * (j + n2 * (i+1))])/(me*nue*nue)*Exy[k + n3 * (j + n2 * (i))])/denominator_e;
        fix[k + n3 * (j + n2 * (i))]=(0.5*(ni[k + n3 * (j + n2 * (i))]+ni[k + n3 * (j + n2 * (i+1))])*mui*Ex[k + n3 * (j + n2 * (i))]-difi*difxni[k + n3 * (j + n2 * (i))]
        -wci*difi*difxyni[k + n3 * (j + n2 * (i))]/nui+wci*q*0.5*(ni[k + n3 * (j + n2 * (i))]+ni[k + n3 * (j + n2 * (i+1))])*Exy[k + n3 * (j + n2 * (i))]/(mi*nui*nui))/denominator_i;
    }


    for ( I = 0; I < n2 * n3; I ++) {
         k = I % n3;
         j = (I - k) / n3;

        if (j * k == 0 ||  j >= jmax-1 || k >= kmax-1) continue;

        if (fex[k + n3 * (j + n2 * (0))] > 0.0){
                fex[k + n3 * (j + n2 * (0))] = 0.0;
                }

        if (fix[k + n3 * (j + n2 * (0))] > 0.0){
                fix[k + n3 * (j + n2 * (0))] = 0.0;
                }

        if (fex[k + n3 * (j + n2 * (imax-2))] < 0.0){
                fex[k + n3 * (j + n2 * (imax-2))] = 0.0;
                }

        if (fix[k + n3 * (j + n2 * (imax-2))] < 0.0){
                fix[k + n3 * (j + n2 * (imax-2))] = 0.0;
                }

    }

// -----------------------------------------------------------------------------------------------

        // Now we calculate the fluxes in z direction
      for ( I = 0; I < N; I ++) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
        if (i * j == 0 || i >= imax-1 || j >= jmax-1 || k >= kmax-1) continue;
        fez[k + n3 * (j + n2 * (i))]=-0.5*(ne[k + n3 * (j + n2 * (i))]+ne[k+1 + n3 * (j + n2 * (i))])*mue*Ez[k + n3 * (j + n2 * (i))]-dife*difzne[k + n3 * (j + n2 * (i))];
        fiz[k + n3 * (j + n2 * (i))]=0.5*(ni[k + n3 * (j + n2 * (i))]+ni[k+1 + n3 * (j + n2 * (i))])*mui*Ez[k + n3 * (j + n2 * (i))]-difi*difzni[k + n3 * (j + n2 * (i))];

    }
       // BC on fluxes

    for ( I = 0; I < n1 * n2; I ++) {
         j = I % n2;
         i = (I - j) / n2;
        if (i * j == 0 || i >= imax-1 || j >= jmax-1) continue;
        if (fez[0 + n3 * (j + n2 * (i))]>0.0){
            fez[0 + n3 * (j + n2 * (i))]=0.0;
        }
        if (fiz[0 + n3 * (j + n2 * (i))]>0.0){
            fiz[0 + n3 * (j + n2 * (i))]=0.0;
        }
        if (fez[kmax-2 + n3 * (j + n2 * (i))]<0.0){
            fez[kmax-2 + n3 * (j + n2 * (i))]=0.0;
        }
        if (fiz[kmax-2 + n3 * (j + n2 * (i))]<0.0){
            fiz[kmax-2 + n3 * (j + n2 * (i))]=0.0;
        }
    }
// -----------------------------------------------------------------------------------------------


       for ( I = 0; I < N; I ++) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
        if (i * j * k == 0 || i >= imax || j >= jmax || k >= kmax) continue;
        ne[k + n3 * (j + n2 * (i))]=ne[k + n3 * (j + n2 * (i))]-dt*(fex[k + n3 * (j + n2 * (i))]-fex[k + n3 * (j + n2 * (i-1))]+fey[k + n3 * (j + n2 * (i))]-fey[k + n3 * (j-1 + n2 * (i))]+fez[k + n3 * (j + n2 * (i))]-fez[k-1 + n3 * (j + n2 * (i))])/h ;
        ni[k + n3 * (j + n2 * (i))]=ni[k + n3 * (j + n2 * (i))]-dt*(fix[k + n3 * (j + n2 * (i))]-fix[k + n3 * (j + n2 * (i-1))]+fiy[k + n3 * (j + n2 * (i))]-fiy[k + n3 * (j-1 + n2 * (i))]+fiz[k + n3 * (j + n2 * (i))]-fiz[k-1 + n3 * (j + n2 * (i))])/h ;
    }





        for ( I = 0; I < n1 * n2; I ++) {
         j = I % n2;
         i = (I - j) / n2;
        if (i * j == 0 || i >= imax || j >= jmax ) continue;

        ne[0 + n3 * (j + n2 * (i))] = -dt*fez[0 + n3 * (j + n2 * (i))]/h ;
        ni[0 + n3 * (j + n2 * (i))] = -dt*fiz[0 + n3 * (j + n2 * (i))]/h ;

    }

     for ( I =0; I < n1 * n3; I ++) {
         k = I % n3;
         i = (I - k) / n3;

        if (i * k == 0 ||  i >= imax || k >= kmax) continue;

        ne[k + n3 * (0 + n2 * (i))] = -dt*fey[k + n3 * (0 + n2 * (i))]/h ;
        ni[k + n3 * (0 + n2 * (i))] = -dt*fiy[k + n3 * (0 + n2 * (i))]/h ;

    }




      for ( I = 0; I < n2 * n3; I ++) {
         k = I % n3;
         j = (I - k) / n3;
        if (j * k == 0 ||  j >= jmax || k >= kmax) continue;

       ne[k + n3 * (j + n2 * (0))]= -dt*fex[k + n3 * (j + n2 * (0))]/h ;
       ni[k + n3 * (j + n2 * (0))]= -dt*fix[k + n3 * (j + n2 * (0))]/h ;


    }




        // BC on densities

     for ( I =0; I < n1 * n3; I ++) {
         k = I % n3;
         i = (I - k) / n3;

        if (i * k == 0 ||  i >= imax || k >= kmax) continue;

        ne[k + n3 * (0 + n2 * (i))] = 0.0 ;
        ni[k + n3 * (0 + n2 * (i))] = 0.0 ;

        ne[k + n3 * (jmax-1 + n2 * (i))] = 0.0 ;
        ni[k + n3 * (jmax-1 + n2 * (i))] = 0.0 ;

    }




      for ( I = 0; I < n2 * n3; I ++) {
         k = I % n3;
         j = (I - k) / n3;
        if (j * k == 0 ||  j >= jmax || k >= kmax) continue;

       ne[k + n3 * (j + n2 * (0))]= 0.0 ;
       ni[k + n3 * (j + n2 * (0))]= 0.0 ;

       ne[k + n3 * (j + n2 * (imax-1))]= 0.0 ;
       ni[k + n3 * (j + n2 * (imax-1))]= 0.0 ;


    }



    for ( I = 0; I < n1*n2; I ++) {
         j = I % n2;
         i = (I - j) / n2;
        if (i * j == 0 || i >= imax+1 || j >= jmax+1) continue;
        ne[kmax-1 + n3 * (j + n2 * (i))]=0.0;
        ne[0 + n3 * (j + n2 * (i))]=0.0;
        ni[kmax-1 + n3 * (j + n2 * (i))]=0.0;
        ni[0 + n3 * (j + n2 * (i))]=0.0;

    }




        // calculating the loss
         sf=0.0;
       for ( I = 0; I < N; I ++) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
         if (i * j * k == 0 || i >= imax || j >= jmax || k >= kmax) continue;
        sf=sf+ne[k + n3 * (j + n2 * (i))] ;
    }

        alpha=(si-sf)/sf;

    for ( I = 0; I < N; I ++) {
         k = I % n3;
         s1 = (I - k) / n3;
         j = s1 % n2;
         i = (s1 - j) / n2;
        if (i * j * k == 0 || i >= imax-1 || j >= jmax-1 || k >= kmax-1) continue;
        ne[k + n3 * (j + n2 * (i))]=ne[k + n3 * (j + n2 * (i))]+alpha*ne[k + n3 * (j + n2 * (i))] ;
        ni[k + n3 * (j + n2 * (i))]=ni[k + n3 * (j + n2 * (i))]+alpha*ne[k + n3 * (j + n2 * (i))] ;
    }
      // if (myTime%100==0.0){
     //  }


     }

}

// -----------------------------------------------------------------------------------------------

int main()
{
int imax = 16, jmax = 16, kmax = 16,i,j,k;
int n1 = imax+3, n2 = jmax+3, n3 = kmax+3;
float qi=1.6E-19,qe=-1.6E-19, kr = 0,ki = 0,si = 0,sf = 0,alpha = 0, q=1.6E-19,pie=3.14159,Ta,w,eps0,Te,Ti,B,Kb,me,mi,nue,nui,denominator_e,denominator_i,nn,dt,h,wce,wci,mue,mui,dife,difi;
int tmax = 40;
float *ne;
float *ni;
float *difxne;
float *difyne;
float *difxni;
float *difyni;
float *difxyne;
float *difxyni;
float *Exy;
float *fexy;
float *fixy;
float *g;
float *g_temp;
float *R;
float *Ex;
float *Ey;
float *fex;
float *fey;
float *fix;
float *fiy;
float *V;
float *L;
float *difzne;
float *difzni;
float *Ez;
float *fez;
float *fiz;
    ne = (float *) malloc(n1 * n2 * n3 * sizeof(float));
    ni = (float *) malloc(n1 * n2 * n3 * sizeof(float));
    difxne = (float *) malloc(n1 * n2 * n3 * sizeof(float));
    difyne = (float *) malloc(n1 * n2 * n3 * sizeof(float));
    difxni = (float *) malloc(n1 * n2 * n3 * sizeof(float));
    difyni = (float *) malloc(n1 * n2 * n3 * sizeof(float));
    difxyne = (float *) malloc(n1 * n2 * n3 * sizeof(float));
    difxyni = (float *) malloc(n1 * n2 * n3 * sizeof(float));
    Exy = (float *) malloc(n1 * n2 * n3 * sizeof(float));
    fexy = (float *) malloc(n1 * n2 * n3 * sizeof(float));
    fixy = (float *) malloc(n1 * n2 * n3 * sizeof(float));
    g = (float *) malloc(n1 * n2 * n3 * sizeof(float));
    g_temp = (float *) malloc(n1 * n2 * n3 * sizeof(float));
    R = (float *) malloc(n1 * n2 * n3 * sizeof(float));
    Ex = (float *) malloc(n1 * n2 * n3 * sizeof(float));
    Ey = (float *) malloc(n1 * n2 * n3 * sizeof(float));
    fex = (float *) malloc(n1 * n2 * n3 * sizeof(float));
    fey = (float *) malloc(n1 * n2 * n3 * sizeof(float));
    fix = (float *) malloc(n1 * n2 * n3 * sizeof(float));
    fiy = (float *) malloc(n1 * n2 * n3 * sizeof(float));
    V = (float *) malloc(n1 * n2 * n3 * sizeof(float));
    L = (float *) malloc(n1 * n2 * n3 * sizeof(float));
    difzne = (float *) malloc(n1 * n2 * n3 * sizeof(float));
    difzni = (float *) malloc(n1 * n2 * n3 * sizeof(float));
    Ez = (float *) malloc(n1 * n2 * n3 * sizeof(float));
    fez = (float *) malloc(n1 * n2 * n3 * sizeof(float));
    fiz = (float *) malloc(n1 * n2 * n3 * sizeof(float));

    Kb    = 1.38E-23;
    B     = 0.0;
    Te    = 2.5*11604.5;
    Ti    = 0.025*11604.5;
    me    = 9.109E-31;
    mi    = 6.633E-26;
    ki    = 0.0;
    dt    = 1.0E-12;
    h     = 1.0E-3;
    eps0  = 8.854E-12;
    si    = 0.0;
    sf    =0.0;



    for ( i=0; i<imax+3;i++){
        for ( j=0; j<jmax+3; j++){
            for ( k=0; k<kmax+3;k++){
                ne[k + n3 * (j + n2 * (i))] = 1e-9;
                ni[k + n3 * (j + n2 * (i))] = 1e-9;
                difxne[k + n3 * (j + n2 * (i))] = 1e-9;
                difyne[k + n3 * (j + n2 * (i))] = 1e-9;
                difxni[k + n3 * (j + n2 * (i))] = 1e-9;
                difyni[k + n3 * (j + n2 * (i))] = 1e-9;
                difxyne[k + n3 * (j + n2 * (i))] = 1e-9;
                difxyni[k + n3 * (j + n2 * (i))] = 1e-9;
                Exy[k + n3 * (j + n2 * (i))] = 1e-9;
                fexy[k + n3 * (j + n2 * (i))] = 1e-9;
                fixy[k + n3 * (j + n2 * (i))] = 1e-9;
                g[k + n3 * (j + n2 * (i))] = 1e-9;
                R[k + n3 * (j + n2 * (i))] = 1e-9;
                Ex[k + n3 * (j + n2 * (i))] = 1e-9;
                Ey[k + n3 * (j + n2 * (i))] = 1e-9;
                fex[k + n3 * (j + n2 * (i))] = 1e-9;
                fey[k + n3 * (j + n2 * (i))] = 1e-9;
                fix[k + n3 * (j + n2 * (i))] = 1e-9;
                fiy[k + n3 * (j + n2 * (i))] = 1e-9;
                V[k + n3 * (j + n2 * (i))] = (rand() % 10)/10.;
                L[k + n3 * (j + n2 * (i))] = 1e-9;
                difzne[k + n3 * (j + n2 * (i))] = 1e-9;
                difzni[k + n3 * (j + n2 * (i))] = 1e-9;
                Ez[k + n3 * (j + n2 * (i))] = 1e-9;
                fez[k + n3 * (j + n2 * (i))] = 1e-9;
                fiz[k + n3 * (j + n2 * (i))] = 1e-9;
             }
        }
    }


    nn=1.33/(Kb*Ti); //neutral density=p/(Kb.T)
    nue=nn*1.1E-19*sqrt(Kb*Te/me); // electron collision frequency= neutral density * sigma_e*Vth_e
    nui=nn*4.4E-19*sqrt(Kb*Ti/mi);
    wce=q*B/me;
    wci=q*B/mi;
    mue=q/(me*nue);
    mui=q/(mi*nui);
    dife=Kb*Te/(me*nue);
    difi=Kb*Ti/(mi*nui);
    ki=0.00002/(nn*dt);
    denominator_e= (1+wce*wce/(nue*nue));
    denominator_i= (1+wci*wci/(nui*nui));
    // Ta and W are just some constants needed for the iterative method that we have used to solve Poisson eq.
    Ta=acos((cos(pie/imax)+cos(pie/jmax)+cos(pie/kmax))/3.0);// needs to be float checked
    w=2.0/(1.0+sin(Ta));
// -----------------------------------------------------------------------------------------------
      //Density initialization
      // To add multiple Gaussian sources, just simply use the density_initialization function at the (x,y) points that you want
        int x_position = 15, y_position = 15, z_position = 15;
       for ( i=1; i<imax-1;i++){
        for ( j=1; j<jmax-1;j++){
            for ( k=1; k<kmax-1;k++){
                ne[k + n3 * (j + n2 * (i))]= 2.0E13;/*
                    1.0E14+1.0E14*exp(-(pow((i-x_position),2)+
                    pow((j-y_position),2)+pow((k-z_position),2))/100.0);*/
                ni[k + n3 * (j + n2 * (i))]=2.0E13;/*
                    1.0E14+1.0E14*exp(-(pow((i-x_position),2)+
                    pow((j-y_position),2)+pow((k-z_position),2))/100.0);*/
            }
        }
    }



        for ( k=1; k<kmax+1; k++) {
          for ( j=1; j<jmax+1; j++) {
           for ( i=1; i<imax+1;i++) {
         si=si+ne[k + n3 * (j + n2 * (i))] ;
          }
         }
        }
// -----------------------------------------------------------------------------------------------

mardas(imax, jmax, kmax, tmax, ne, ni, difxne, difyne, difxni, difyni, difxyne, difxyni, Exy, fexy, fixy, g, g_temp, R, Ex, Ey, fex, fey, fix, fiy, V, L, difzne, difzni, Ez, fez, fiz, qi, qe, kr, ki, si, sf, alpha, q, pie,Ta ,w ,eps0 , Te, Ti, B, Kb, me, mi, nue, nui, denominator_e, denominator_i, nn, dt, h, wce, wci, mue, mui, dife, difi);




    printf("%f\n", V[5 + (kmax+3) * (5 + (jmax+3) * 5)]);
 //   printf("%f\n", V[my_index + (kmax+3) * (my_index + (jmax+3) * my_index)]);

}

